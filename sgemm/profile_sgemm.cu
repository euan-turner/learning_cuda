#include <iostream>
#include <stdio.h>
#include <nvToolsExt.h>

#include "sgemm.cuh"
#include "naive_sgemm.cuh"
#include "coalesced_sgemm.cuh"

#define CEIL_DIV(x, y) (((x) + (y) - 1) / (y))
#define CUDA_CHECK(call) { \
  hipError_t err = call; \
  if (err != hipSuccess) { \
    fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
    exit(EXIT_FAILURE); \
  } \
}
// TODO: Use this

const int BLOCK_SIZE = 32;

/**

 * @brief Profile a kernel function for SGEMM (Single-precision General Matrix Multiply)
 * 
 *
 * Each kernel computes \f$ C = \alpha AB + \beta C \f$
  * where \f$ A \in \mathbb{R}^{M \times K} \f$, \f$ B \in \mathbb{R}^{K \times N} \f$, and \f$ C \in \mathbb{R}^{M \times N} \f$.
 */
void profileKernel(
  void (*kernel)(SgemmParams),
  const std::string& kernel_name,
  const dim3 dimGrid,
  const dim3 dimBlock,
  SgemmParams ps,
  const int warmup_runs = 10,
  const int test_runs = 100
) {
  float* C_original;
  hipMalloc(&C_original, ps.M * ps.N * sizeof(float));
  hipMemcpy(C_original, ps.C, ps.M * ps.N * sizeof(float), hipMemcpyDeviceToDevice);

  nvtxRangePushA((kernel_name + " Warm-Up Phase").c_str());
  for (int i = 0; i < warmup_runs; i++) {
      // Reset C before each run
      hipMemcpy(ps.C, C_original, ps.M * ps.N * sizeof(float), hipMemcpyDeviceToDevice);
      kernel<<<dimGrid, dimBlock>>>(ps);
  }
  hipDeviceSynchronize();
  nvtxRangePop();

  nvtxRangePushA((kernel_name + " Test Phase").c_str());
  for (int i = 0; i < test_runs; i++) {
      // Reset C before each run
      hipMemcpy(ps.C, C_original, ps.M * ps.N * sizeof(float), hipMemcpyDeviceToDevice);
      kernel<<<dimGrid, dimBlock>>>(ps);
  }
  hipDeviceSynchronize();
  nvtxRangePop();

  hipFree(C_original);
}

int main(void) {
  const int M = 512;
  const int N = 512;
  const int K = 512;
  const size_t matrix_size = M * N * sizeof(float); 

  float *A, *B, *C;

  hipMallocManaged(&A, matrix_size);
  hipMallocManaged(&B, matrix_size);
  hipMallocManaged(&C, matrix_size);

  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      A[i*N + j] = i * N + j;
      B[i*N + j] = i * N + j;
      C[i*N + j] = 0;
    }
  }

  SgemmParams params = {.M = M, .N = N, .K = K, .alpha = 1.0f, .beta = 0.0f, .A = A, .B = B, .C = C};
  
  profileKernel(naive_sgemm, "Naive SGEMM Kernel", dim3(CEIL_DIV(M, BLOCK_SIZE), CEIL_DIV(N, BLOCK_SIZE), 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1), 
    params);
  std::cout << "SGEMM Naive Kernel executed" << std::endl;

  profileKernel(
    coalesced_sgemm<BLOCK_SIZE>,
    "Coalesced SGEMM Kernel", dim3(CEIL_DIV(M, BLOCK_SIZE), CEIL_DIV(N, BLOCK_SIZE), 1), dim3(BLOCK_SIZE * BLOCK_SIZE, 1, 1), 
    params);
  std::cout << "SGEMM Coalesced Kernel executed" << std::endl;

  hipFree(A);
  hipFree(B);
  hipFree(C);

}