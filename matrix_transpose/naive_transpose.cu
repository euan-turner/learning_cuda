#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <nvToolsExt.h>

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

__global__
void copy(float *odata, const float *idata) {
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    odata[(y+j)*width + x] = idata[(y+j)*width + x];
  }
}

__global__
void transposeNaive(float *odata, const float *idata) {
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
  }
}

__global__
void transposeCoalesced(float *odata, const float *idata) {
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;
  
  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*gridDim.x*TILE_DIM + x];
  }

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
  }
}

__global__
void transposeCoalescedNoBankConflicts(float *odata, const float *idata) {
  __shared__ float tile[TILE_DIM][TILE_DIM + 1];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;
  
  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*gridDim.x*TILE_DIM + x];
  }

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
  }
}

void profileKernel(
  void (*kernel)(float*, const float*),
  const std::string& kernel_name,
  float *odata,
  const float *idata,
  const int DIM,
  const int warmup_runs = 10,
  const int test_runs = 100
) {
  dim3 dimGrid(DIM/TILE_DIM, DIM/TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

  nvtxRangePushA((kernel_name + " Warm-Up Phase").c_str());
  for (int i = 0; i < warmup_runs; i++) {
    kernel<<<dimGrid, dimBlock>>>(odata, idata);
  }
  hipDeviceSynchronize();
  nvtxRangePop();
  nvtxRangePushA((kernel_name + " Test Phase").c_str());
  for (int i = 0; i < test_runs; i++) {
    kernel<<<dimGrid, dimBlock>>>(odata, idata);
  }
  hipDeviceSynchronize();
  nvtxRangePop();
}

int main(void) {
  const int DIM = 512;
  const size_t matrix_size = DIM * DIM * sizeof(float); 

  float *src, *dst;

  hipMallocManaged(&src, matrix_size);
  hipMallocManaged(&dst, matrix_size);

  for (int i = 0; i < DIM; i++) {
    for (int j = 0; j < DIM; j++) {
      src[i*DIM + j] = i * DIM + j;
      dst[i*DIM + j] = 0;
    }
  }

  profileKernel(copy, "Copy Kernel", dst, src, DIM);
  std::cout << "Copy Kernel executed" << std::endl;
  profileKernel(transposeNaive, "Naive Transpose Kernel", dst, src, DIM);
  std::cout << "Naive Transpose Kernel executed" << std::endl;
  profileKernel(transposeCoalesced, "Coalesced Transpose Kernel", dst, src, DIM);
  std::cout << "Coalesced Transpose Kernel executed" << std::endl;
  profileKernel(transposeCoalescedNoBankConflicts, "Coalesced No Bank Conflicts Transpose Kernel", dst, src, DIM);
  std::cout << "Coalesced No Bank Conflicts Transpose Kernel executed" << std::endl;

  hipFree(src);
  hipFree(dst);

  return 0;
}